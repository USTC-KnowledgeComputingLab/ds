#include "hip/hip_runtime.h"
#include "config.h++"
#include "cuda_compact.h++"
#include "helper.h++"
#include "item.h++"
#include "list.h++"
#include "rule.h++"
#include "string.h++"
#include "term.h++"
#include "utility.h++"
#include "variable.h++"

#include <chrono>
#include <cstdio>
#include <functional>
#include <iostream>
#include <map>
#include <set>

// TODO: check out of bound
// TODO: manual stack
// TODO: container on CUDA
// TODO: unification algorithm need to be updated

/// @brief 验证两个rule是否相等
/// @param rule_1 第一个rule
/// @param rule_2 第二个rule
/// @return 相等则返回true, 否则是false
CUDA_HOST_DEVICE bool rule_equal(cuds::rule_t* rule_1, cuds::rule_t* rule_2) {
    if (rule_1->data_size() != rule_2->data_size()) {
        return false;
    }
    auto data_size = rule_1->data_size();
    if (cuds::memcmp(rule_1, rule_2, data_size) != 0) {
        return false;
    }
    return true;
}

enum class match_flag_t {
    null = 0,
    fail = 1,
    rule = 2,
    fact = 3,
    found = 4
};

struct match_result_t {
    /// @brief 包含了成功与否, 以及是fact还是rule等信息
    match_flag_t flag;
    /// @brief 结果rule的大小, 由于有时候也用来顺便存储指标, 所以不是length_t而是更大的size_t
    std::size_t size;
};

/// @brief 将rule和fact进行匹配的核函数
/// @param match_result 匹配的结果, 包含成功与否, 产出对象大小等信息
/// @param rule_result 产出对象
/// @param result_size_threshold 产出对象的截断大小
/// @param rule 用于匹配的rule
/// @param fact 用于匹配的fact
/// @param rules 用于去重的rules合集
/// @param rules_size 用于去重的rules合集大小
/// @param facts 用于去重的facts合集
/// @param facts_size 用于去重的facts合集大小
/// @param target 用于查询是否成功的目标对象
__device__ void match(
    match_result_t* match_result,
    cuds::rule_t* rule_result,
    cuds::length_t result_size_threshold,
    cuds::rule_t* rule,
    cuds::rule_t* fact,
    cuds::rule_t** rules,
    std::size_t rules_size,
    cuds::rule_t** facts,
    std::size_t facts_size,
    cuds::rule_t* target
) {
    rule_result->match(rule, fact);
    if (!rule_result->valid()) {
        match_result->flag = match_flag_t::fail;
        return;
    }
    if (rule_result->data_size() > result_size_threshold) {
        match_result->flag = match_flag_t::fail;
        return;
    }
    if (rule_result->premises_count() != 0) {
        // rule
        for (std::size_t rule_index = 0; rule_index < rules_size; ++rule_index) {
            cuds::rule_t* old_rule = rules[rule_index];
            if (rule_equal(old_rule, rule_result)) {
                match_result->flag = match_flag_t::fail;
                return;
            }
        }
        match_result->flag = match_flag_t::rule;
        match_result->size = rule_result->data_size();
    } else {
        // fact
        for (std::size_t fact_index = 0; fact_index < facts_size; ++fact_index) {
            cuds::rule_t* old_fact = facts[fact_index];
            if (rule_equal(old_fact, rule_result)) {
                match_result->flag = match_flag_t::fail;
                return;
            }
        }
        match_result->flag = match_flag_t::fact;
        match_result->size = rule_result->data_size();
        if (rule_equal(target, rule_result)) {
            match_result->flag = match_flag_t::found;
        }
    }
}

/// @brief 将rule和fact进行批量匹配的核函数接口
/// @param match_result_pool 匹配结果池
/// @param rule_result_pool 产出对象池
/// @param single_result_size 产出对象池中单个对象的大小
/// @param single_result_size_threshold 产出对象的截断大小
/// @param local_index_offset 单次调用的任务指标偏移
/// @param rules 用于去重的rules合集
/// @param old_old_rules_size 老rules合集的大小
/// @param old_rules_size 匹配前rules合集的大小
/// @param facts 用于去重的facts合集
/// @param old_old_facts_size 老facts合集的大小
/// @param old_facts_size 匹配前facts合集的大小
/// @param target 用于查询是否成功的目标对象
__global__ void process(
    match_result_t* match_result_pool,
    cuds::rule_t* rule_result_pool,
    cuds::length_t single_result_size,
    cuds::length_t single_result_size_threshold,
    std::size_t local_index_offset,
    cuds::rule_t** rules,
    std::size_t old_old_rules_size,
    std::size_t old_rules_size,
    cuds::rule_t** facts,
    std::size_t old_old_facts_size,
    std::size_t old_facts_size,
    cuds::rule_t* target
) {
    std::size_t local_thread_count = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    std::size_t local_thread_index =
        (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.z * blockDim.y * blockDim.x) +
        (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);
    std::size_t global_job_count = old_rules_size * old_facts_size - old_old_rules_size * old_old_facts_size;
    std::size_t global_job_index = local_thread_index + local_index_offset;
    std::size_t remained_job_count = global_job_count - local_index_offset;
    std::size_t local_job_count = local_thread_count > remained_job_count ? remained_job_count : local_thread_count;

    if (global_job_index >= global_job_count) {
        return;
    }

    if (local_thread_index == 0) {
        match_result_pool[local_job_count].flag = match_flag_t::null;
    }

    std::size_t rule_index;
    std::size_t fact_index;
    if (global_job_index < (old_rules_size - old_old_rules_size) * old_facts_size) {
        rule_index = global_job_index / old_facts_size + old_old_rules_size;
        fact_index = global_job_index % old_facts_size;
    } else {
        std::size_t temp_index = global_job_index - (old_rules_size - old_old_rules_size) * old_facts_size;
        rule_index = temp_index % old_old_rules_size;
        fact_index = temp_index / old_old_rules_size + old_old_facts_size;
    }

    match_result_t* match_result = &match_result_pool[local_thread_index];
    cuds::rule_t* rule_result = cuds::with_offset(rule_result_pool, local_thread_index * single_result_size);
    match(
        match_result,
        rule_result,
        single_result_size_threshold,
        rules[rule_index],
        facts[fact_index],
        rules,
        old_rules_size,
        facts,
        old_facts_size,
        target
    );

    if (match_result->flag == match_flag_t::found) {
        match_result_pool[local_job_count].flag = match_flag_t::found;
        match_result_pool[local_job_count].size = local_thread_index;
    }
}

void run() {
    int temp_data_size = 1000;
    int temp_text_size = 1000;
    int single_result_size = 30000;
    int single_result_size_threshold = 2000;
    int cuda_stack_size = 2000;
    int max_block_per_call = 4000;
    int thread_per_block = 16;

    CHECK_CUDA_ERROR(hipDeviceSetLimit(hipLimitStackSize, cuda_stack_size));

    // P -> Q, P |- Q
    auto mp = cuds::text_to_rule(
        "('P -> 'Q)\n"
        "'P\n"
        "----------\n"
        "'Q",
        temp_data_size
    );
    // p -> (q -> p)
    auto axiom1 = cuds::text_to_rule(
        "------------------\n"
        "('p -> ('q -> 'p))\n",
        temp_data_size
    );
    // (p -> (q -> r)) -> ((p -> q) -> (p -> r))
    auto axiom2 = cuds::text_to_rule(
        "--------------------------------------------------\n"
        "(('p -> ('q -> 'r)) -> (('p -> 'q) -> ('p -> 'r)))\n",
        temp_data_size
    );
    // (!p -> !q) -> (q -> p)
    auto axiom3 = cuds::text_to_rule(
        "----------------------------------\n"
        "(((! 'p) -> (! 'q)) -> ('q -> 'p))\n",
        temp_data_size
    );

    auto premise = cuds::text_to_rule("(! (! X))", temp_data_size);
    auto target = cuds::text_to_rule("X", temp_data_size);

    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> rules;
    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> facts;
    rules.push_back(copy_host_to_device(mp));
    facts.push_back(copy_host_to_device(axiom1));
    facts.push_back(copy_host_to_device(axiom2));
    facts.push_back(copy_host_to_device(axiom3));
    facts.push_back(copy_host_to_device(premise));
    cuds::unique_cuda_malloc_ptr<cuds::rule_t> target_d = copy_host_to_device(target);

    double kernel_time = 0;

    std::size_t old_old_rules_size = 0;
    std::size_t old_old_facts_size = 0;
    while (true) {
        std::size_t old_rules_size = rules.size();
        std::size_t old_facts_size = facts.size();

        cuds::rule_t** rules_d;
        cuds::rule_t** facts_d;
        CHECK_CUDA_ERROR(hipMalloc(&rules_d, old_rules_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(rules_d, rules.data(), old_rules_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMalloc(&facts_d, old_facts_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(facts_d, facts.data(), old_facts_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));

        std::size_t global_job_count = old_rules_size * old_facts_size - old_old_rules_size * old_old_facts_size;
        std::size_t max_thread_per_call = max_block_per_call * thread_per_block;
        for (std::size_t local_index_offset = 0; local_index_offset < global_job_count; local_index_offset += max_thread_per_call) {
            printf(".");
            std::size_t remained_job_count = global_job_count - local_index_offset;
            std::size_t local_job_count = remained_job_count < max_thread_per_call ? remained_job_count : max_thread_per_call;
            std::size_t block_count = (local_job_count + thread_per_block - 1) / thread_per_block;

            match_result_t* match_result_pool_d;
            CHECK_CUDA_ERROR(hipMalloc(&match_result_pool_d, sizeof(match_result_t) * (local_job_count + 1)));
            cuds::rule_t* rule_result_pool_d;
            CHECK_CUDA_ERROR(hipMalloc(&rule_result_pool_d, single_result_size * local_job_count));

            auto start = std::chrono::high_resolution_clock::now();
            process<<<block_count, thread_per_block>>>(
                match_result_pool_d,
                rule_result_pool_d,
                single_result_size,
                single_result_size_threshold,
                local_index_offset,
                rules_d,
                old_old_rules_size,
                old_rules_size,
                facts_d,
                old_old_facts_size,
                old_facts_size,
                target_d.get()
            );
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> duration = end - start;
            kernel_time += duration.count();

            match_result_t* match_result_pool_h = reinterpret_cast<match_result_t*>(malloc(sizeof(match_result_t) * (local_job_count + 1)));
            CHECK_CUDA_ERROR(
                hipMemcpy(match_result_pool_h, match_result_pool_d, sizeof(match_result_t) * (local_job_count + 1), hipMemcpyDeviceToHost)
            );

            if (match_result_pool_h[local_job_count].flag == match_flag_t::found) {
                std::size_t job_index = match_result_pool_h[local_job_count].size;
                match_result_t* match_result_h = &match_result_pool_h[job_index];
                cuds::rule_t* result_n = reinterpret_cast<cuds::rule_t*>(malloc(match_result_h->size));
                CHECK_CUDA_ERROR(hipMemcpy(
                    result_n,
                    cuds::with_offset(rule_result_pool_d, single_result_size * job_index),
                    match_result_h->size,
                    hipMemcpyDeviceToHost
                ));
                char* text_n = reinterpret_cast<char*>(malloc(temp_text_size));
                *(result_n->print(text_n)) = 0;
                printf("Found!\n");
                printf("%s", text_n);
                printf("Last job count: %d\n", global_job_count);
                printf("Kernel time: %lf seconds\n", kernel_time);
                free(text_n);
                free(result_n);
                free(match_result_pool_h);
                CHECK_CUDA_ERROR(hipFree(rule_result_pool_d));
                CHECK_CUDA_ERROR(hipFree(match_result_pool_d));
                CHECK_CUDA_ERROR(hipFree(rules_d));
                CHECK_CUDA_ERROR(hipFree(facts_d));
                return;
            }

            for (int job_index = 0; job_index < local_job_count; ++job_index) {
                match_result_t* match_result_h = &match_result_pool_h[job_index];
                if (match_result_h->flag == match_flag_t::fact || match_result_h->flag == match_flag_t::rule) {
                    cuds::rule_t* result_n;
                    CHECK_CUDA_ERROR(hipMalloc(&result_n, match_result_h->size));
                    CHECK_CUDA_ERROR(hipMemcpy(
                        result_n,
                        cuds::with_offset(rule_result_pool_d, single_result_size * job_index),
                        match_result_h->size,
                        hipMemcpyDeviceToDevice
                    ));

                    if (match_result_h->flag == match_flag_t::rule) {
                        // rule
                        rules.emplace_back(result_n);
                    } else {
                        // fact
                        facts.emplace_back(result_n);
                    }
                }
            }

            free(match_result_pool_h);
            CHECK_CUDA_ERROR(hipFree(rule_result_pool_d));
            CHECK_CUDA_ERROR(hipFree(match_result_pool_d));
        }

        CHECK_CUDA_ERROR(hipFree(rules_d));
        CHECK_CUDA_ERROR(hipFree(facts_d));

        old_old_rules_size = old_rules_size;
        old_old_facts_size = old_facts_size;

        printf("\n");
    }
}

void timer(std::function<void()> func) {
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds\n" << std::flush;
}

int main() {
    for (auto i = 0; i < 10; ++i) {
        timer(run);
    }
}
