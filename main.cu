#include "hip/hip_runtime.h"
#include "config.h++"
#include "cuda_compact.h++"
#include "helper.h++"
#include "item.h++"
#include "list.h++"
#include "string.h++"
#include "term.h++"
#include "utility.h++"
#include "variable.h++"

#include <chrono>
#include <cstdio>
#include <functional>
#include <iostream>
#include <map>
#include <set>

// TODO: check out of bound
// TODO: manual stack
// TODO: container on CUDA

CUDA_HOST_DEVICE bool rule_equal(cuds::rule_t* rule_1, cuds::rule_t* rule_2) {
    if (rule_1->data_size() != rule_2->data_size()) {
        return false;
    }
    auto data_size = rule_1->data_size();
    if (cuds::memcmp(rule_1, rule_2, data_size) != 0) {
        return false;
    }
    return true;
}

enum class match_flag_t {
    null = 0,
    fail = 1,
    rule = 2,
    fact = 3,
    found = 4
};

struct match_result_t {
    match_flag_t flag;
    cuds::length_t size;
};

__device__ void match(
    match_result_t* match_result,
    cuds::rule_t* rule_result,
    cuds::length_t result_size_threshold,
    cuds::rule_t* rule,
    cuds::rule_t* fact,
    cuds::rule_t** rules,
    cuds::length_t rules_size,
    cuds::rule_t** facts,
    cuds::length_t facts_size,
    cuds::rule_t* target
) {
    rule_result->match(rule, fact);
    if (!rule_result->valid()) {
        match_result->flag = match_flag_t::fail;
        return;
    }
    if (rule_result->data_size() > result_size_threshold) {
        match_result->flag = match_flag_t::fail;
        return;
    }
    if (rule_result->premises_count() != 0) {
        // rule
        for (cuds::length_t rule_index = 0; rule_index < rules_size; ++rule_index) {
            cuds::rule_t* old_rule = rules[rule_index];
            if (rule_equal(old_rule, rule_result)) {
                match_result->flag = match_flag_t::fail;
                return;
            }
        }
        match_result->flag = match_flag_t::rule;
        match_result->size = rule_result->data_size();
    } else {
        // fact
        for (cuds::length_t fact_index = 0; fact_index < facts_size; ++fact_index) {
            cuds::rule_t* old_fact = facts[fact_index];
            if (rule_equal(old_fact, rule_result)) {
                match_result->flag = match_flag_t::fail;
                return;
            }
        }
        match_result->flag = match_flag_t::fact;
        match_result->size = rule_result->data_size();
        if (rule_equal(target, rule_result)) {
            match_result->flag = match_flag_t::found;
        }
    }
}

__global__ void process(
    match_result_t* match_result_pool,
    cuds::rule_t* rule_result_pool,
    cuds::length_t single_result_size,
    cuds::length_t single_result_size_threshold,
    cuds::rule_t** rules,
    cuds::length_t old_old_rules_size,
    cuds::length_t old_rules_size,
    cuds::rule_t** facts,
    cuds::length_t old_old_facts_size,
    cuds::length_t old_facts_size,
    cuds::rule_t* target
) {
    int thread_count = int(old_rules_size) * int(old_facts_size) - int(old_old_rules_size) * int(old_old_facts_size);
    int thread_index = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.z * blockDim.y * blockDim.x) +
                       (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);

    if (thread_index >= thread_count) {
        return;
    }

    int rule_index;
    int fact_index;
    if (thread_index < (old_rules_size - old_old_rules_size) * old_facts_size) {
        rule_index = thread_index / old_facts_size + old_old_rules_size;
        fact_index = thread_index % old_facts_size;
    } else {
        int temp_index = thread_index - (old_rules_size - old_old_rules_size) * old_facts_size;
        rule_index = temp_index % old_old_rules_size;
        fact_index = temp_index / old_old_rules_size + old_old_facts_size;
    }

    if (thread_index == 0) {
        match_result_pool[thread_count].flag = match_flag_t::null;
    }

    match_result_t* match_result = &match_result_pool[thread_index];
    cuds::rule_t* rule_result = cuds::with_offset(rule_result_pool, thread_index * single_result_size);
    match(
        match_result,
        rule_result,
        single_result_size_threshold,
        rules[rule_index],
        facts[fact_index],
        rules,
        old_rules_size,
        facts,
        old_facts_size,
        target
    );

    if (match_result->flag == match_flag_t::found) {
        match_result_pool[thread_count].flag = match_flag_t::found;
    }
}

void run() {
    int single_result_size = 32000;
    int single_result_size_threshold = 80;
    int cuda_stack_size = 2000;
    int thread_per_block = 32;

    CHECK_CUDA_ERROR(hipDeviceSetLimit(hipLimitStackSize, cuda_stack_size));

    // P -> Q, P |- Q
    auto mp = cuds::text_to_rule(
        "('P -> 'Q)\n"
        "'P\n"
        "----------\n"
        "'Q",
        1000
    );
    // p -> (q -> p)
    auto axiom1 = cuds::text_to_rule(
        "------------------\n"
        "('p -> ('q -> 'p))\n",
        1000
    );
    // (p -> (q -> r)) -> ((p -> q) -> (p -> r))
    auto axiom2 = cuds::text_to_rule(
        "--------------------------------------------------\n"
        "(('p -> ('q -> 'r)) -> (('p -> 'q) -> ('p -> 'r)))\n",
        1000
    );
    // (!p -> !q) -> (q -> p)
    auto axiom3 = cuds::text_to_rule(
        "----------------------------------\n"
        "(((! 'p) -> (! 'q)) -> ('q -> 'p))\n",
        1000
    );

    auto premise = cuds::text_to_rule("(! (! X))", 1000);
    auto target = cuds::text_to_rule("X", 1000);

    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> rules;
    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> facts;
    rules.push_back(copy_host_to_device(mp));
    facts.push_back(copy_host_to_device(axiom1));
    facts.push_back(copy_host_to_device(axiom2));
    facts.push_back(copy_host_to_device(axiom3));
    facts.push_back(copy_host_to_device(premise));
    cuds::unique_cuda_malloc_ptr<cuds::rule_t> target_d = copy_host_to_device(target);

    double kernel_time;

    cuds::length_t old_old_rules_size = 0;
    cuds::length_t old_old_facts_size = 0;
    while (true) {
        cuds::length_t old_rules_size = rules.size();
        cuds::length_t old_facts_size = facts.size();

        cuds::rule_t** rules_d;
        cuds::rule_t** facts_d;
        CHECK_CUDA_ERROR(hipMalloc(&rules_d, old_rules_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(rules_d, rules.data(), old_rules_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMalloc(&facts_d, old_facts_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(facts_d, facts.data(), old_facts_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));

        int thread_count = int(old_rules_size) * int(old_facts_size) - int(old_old_rules_size) * int(old_old_facts_size);

        match_result_t* match_result_pool_d;
        CHECK_CUDA_ERROR(hipMalloc(&match_result_pool_d, sizeof(match_result_t) * (thread_count + 1)));
        cuds::rule_t* result_pool_d;
        CHECK_CUDA_ERROR(hipMalloc(&result_pool_d, single_result_size * thread_count));

        auto start = std::chrono::high_resolution_clock::now();
        process<<<(thread_count + thread_per_block - 1) / thread_per_block, thread_per_block>>>(
            match_result_pool_d,
            result_pool_d,
            single_result_size,
            single_result_size_threshold,
            rules_d,
            old_old_rules_size,
            old_rules_size,
            facts_d,
            old_old_facts_size,
            old_facts_size,
            target_d.get()
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        kernel_time += duration.count();

        match_result_t* match_result_pool_h = reinterpret_cast<match_result_t*>(malloc(sizeof(match_result_t) * (thread_count + 1)));
        CHECK_CUDA_ERROR(hipMemcpy(match_result_pool_h, match_result_pool_d, sizeof(match_result_t) * (thread_count + 1), hipMemcpyDeviceToHost));

        if (match_result_pool_h[thread_count].flag == match_flag_t::found) {
            printf("Found!\n");
            printf("Kernel time: %lf seconds\n", kernel_time);
            printf("Last thread count: %d\n", thread_count);
            return;
        }

        for (int thread_index = 0; thread_index < thread_count; ++thread_index) {
            match_result_t* match_result_h = &match_result_pool_h[thread_index];
            if (match_result_h->flag == match_flag_t::fact || match_result_h->flag == match_flag_t::rule) {
                cuds::rule_t* result_n;
                CHECK_CUDA_ERROR(hipMalloc(&result_n, match_result_h->size));
                CHECK_CUDA_ERROR(hipMemcpy(
                    result_n,
                    cuds::with_offset(result_pool_d, single_result_size * thread_index),
                    match_result_h->size,
                    hipMemcpyDeviceToDevice
                ));

                if (match_result_h->flag == match_flag_t::rule) {
                    // rule
                    rules.emplace_back(result_n);
                } else {
                    // fact
                    facts.emplace_back(result_n);
                }
            }
        }

        CHECK_CUDA_ERROR(hipFree(match_result_pool_d));

        CHECK_CUDA_ERROR(hipFree(rules_d));
        CHECK_CUDA_ERROR(hipFree(facts_d));

        old_old_rules_size = old_rules_size;
        old_old_facts_size = old_facts_size;
    }
}

void timer(std::function<void()> func) {
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds\n" << std::flush;
}

int main() {
    timer(run);
    timer(run);
    timer(run);
    timer(run);
}
