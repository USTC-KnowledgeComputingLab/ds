#include "hip/hip_runtime.h"
#include "config.h++"
#include "cuda_compact.h++"
#include "item.h++"
#include "list.h++"
#include "string.h++"
#include "term.h++"
#include "utility.h++"
#include "variable.h++"

#include <chrono>
#include <cstdio>
#include <functional>
#include <iostream>
#include <map>
#include <set>

// TODO: check out of bound
// TODO: manual stack
// TODO: container on CUDA

CUDA_HOST_DEVICE bool rule_equal(cuds::rule_t* rule_1, cuds::rule_t* rule_2) {
    if (rule_1->data_size() != rule_2->data_size()) {
        return false;
    }
    auto data_size = rule_1->data_size();
    if (cuds::memcmp(rule_1, rule_2, data_size) != 0) {
        return false;
    }
    return true;
}

enum class match_flag_t {
    null = 0,
    fail = 1,
    rule = 2,
    fact = 3,
    found = 4
};

__global__ void match(
    match_flag_t* flag,
    cuds::length_t* size,
    cuds::rule_t* result,
    cuds::length_t threshold,
    cuds::rule_t* rule_1,
    cuds::rule_t* rule_2,
    cuds::rule_t** rules,
    cuds::length_t rules_size,
    cuds::rule_t** facts,
    cuds::length_t facts_size,
    cuds::rule_t* target
) {
    result->match(rule_1, rule_2);
    if (!result->valid()) {
        *flag = match_flag_t::fail;
        return;
    }
    if (result->data_size() > threshold) {
        *flag = match_flag_t::fail;
        return;
    }
    if (result->premises_count() != 0) {
        // rule
        for (cuds::length_t rule_index = 0; rule_index < rules_size; ++rule_index) {
            cuds::rule_t* old_rule = rules[rule_index];
            if (rule_equal(old_rule, result)) {
                *flag = match_flag_t::fail;
                return;
            }
        }
        *flag = match_flag_t::rule;
        *size = result->data_size();
    } else {
        // fact
        for (cuds::length_t fact_index = 0; fact_index < facts_size; ++fact_index) {
            cuds::rule_t* old_fact = facts[fact_index];
            if (rule_equal(old_fact, result)) {
                *flag = match_flag_t::fail;
                return;
            }
        }
        *flag = match_flag_t::fact;
        *size = result->data_size();
        if (rule_equal(target, result)) {
            *flag = match_flag_t::found;
        }
    }
}

void run() {
    hipDeviceSetLimit(hipLimitStackSize, 2000);

    // P -> Q, P |- Q
    auto mp = cuds::text_to_rule(
        "('P -> 'Q)\n"
        "'P\n"
        "----------\n"
        "'Q",
        1000
    );
    // p -> (q -> p)
    auto axiom1 = cuds::text_to_rule(
        "------------------\n"
        "('p -> ('q -> 'p))\n",
        1000
    );
    // (p -> (q -> r)) -> ((p -> q) -> (p -> r))
    auto axiom2 = cuds::text_to_rule(
        "--------------------------------------------------\n"
        "(('p -> ('q -> 'r)) -> (('p -> 'q) -> ('p -> 'r)))\n",
        1000
    );
    // (!p -> !q) -> (q -> p)
    auto axiom3 = cuds::text_to_rule(
        "----------------------------------\n"
        "(((! 'p) -> (! 'q)) -> ('q -> 'p))\n",
        1000
    );

    auto premise = cuds::text_to_rule("(! (! X))", 1000);
    auto target = cuds::text_to_rule("X", 1000);

    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> rules;
    std::vector<cuds::unique_cuda_malloc_ptr<cuds::rule_t>> facts;
    rules.push_back(copy_host_to_device(mp));
    facts.push_back(copy_host_to_device(axiom1));
    facts.push_back(copy_host_to_device(axiom2));
    facts.push_back(copy_host_to_device(axiom3));
    facts.push_back(copy_host_to_device(premise));
    cuds::unique_cuda_malloc_ptr<cuds::rule_t> target_d = copy_host_to_device(target);

    cuds::length_t old_old_rules_size = 0;
    cuds::length_t old_old_facts_size = 0;
    while (true) {
        cuds::length_t old_rules_size = rules.size();
        cuds::length_t old_facts_size = facts.size();

        cuds::rule_t** device_rules;
        cuds::rule_t** device_facts;
        CHECK_CUDA_ERROR(hipMalloc(&device_rules, old_rules_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(device_rules, rules.data(), old_rules_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMalloc(&device_facts, old_facts_size * sizeof(cuds::rule_t*)));
        CHECK_CUDA_ERROR(hipMemcpy(device_facts, facts.data(), old_facts_size * sizeof(cuds::rule_t*), hipMemcpyHostToDevice));

        for (cuds::length_t rule_index = 0; rule_index < old_rules_size; ++rule_index) {
            for (cuds::length_t fact_index = 0; fact_index < old_facts_size; ++fact_index) {
                if (rule_index < old_old_rules_size && fact_index < old_old_facts_size) {
                    continue;
                }

                cuds::rule_t* result_d;
                CHECK_CUDA_ERROR(hipMalloc(&result_d, 32000));
                match_flag_t* flag_d;
                CHECK_CUDA_ERROR(hipMalloc(&flag_d, sizeof(match_flag_t)));
                cuds::length_t* size_d;
                CHECK_CUDA_ERROR(hipMalloc(&size_d, sizeof(cuds::length_t)));

                match<<<1, 1>>>(
                    flag_d,
                    size_d,
                    result_d,
                    80,
                    rules[rule_index].get(),
                    facts[fact_index].get(),
                    device_rules,
                    old_rules_size,
                    device_facts,
                    old_facts_size,
                    target_d.get()
                );

                cuds::length_t size_h;
                hipMemcpy(&size_h, size_d, sizeof(cuds::length_t), hipMemcpyDeviceToHost);
                match_flag_t flag_h;
                hipMemcpy(&flag_h, flag_d, sizeof(match_flag_t), hipMemcpyDeviceToHost);

                CHECK_CUDA_ERROR(hipFree(size_d));
                CHECK_CUDA_ERROR(hipFree(flag_d));
                if (flag_h == match_flag_t::found) {
                    printf("Found!\n");
                    CHECK_CUDA_ERROR(hipFree(result_d));
                    return;
                }
                if (flag_h == match_flag_t::fact || flag_h == match_flag_t::rule) {
                    cuds::rule_t* result_n;
                    CHECK_CUDA_ERROR(hipMalloc(&result_n, size_h));
                    CHECK_CUDA_ERROR(hipMemcpy(result_n, result_d, size_h, hipMemcpyDeviceToDevice));

                    if (flag_h == match_flag_t::rule) {
                        // rule
                        rules.emplace_back(result_n);
                    } else {
                        // fact
                        facts.emplace_back(result_n);
                    }
                }
                CHECK_CUDA_ERROR(hipFree(result_d));
            }
        }

        CHECK_CUDA_ERROR(hipFree(device_rules));
        CHECK_CUDA_ERROR(hipFree(device_facts));

        old_old_rules_size = old_rules_size;
        old_old_facts_size = old_facts_size;
    }
}

void timer(std::function<void()> func) {
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds\n" << std::flush;
}

int main() {
    timer(run);
    std::cout << "Run again...\n" << std::flush;
    timer(run);
}
